#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <stdlib.h>
#include <stdio.h>
#include <stdio.h>

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

hipError_t addWithCuda(int* c, const int* a, const int* b, unsigned int size);

#define N 10000000
#define MAX_ERR 1e-6

__global__ void vector_add(float* out, float* a, float* b, int n) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    while (tid < N) {
        out[tid] = a[tid] + b[tid];
        tid += blockDim.x * gridDim.x;
    }
    
    
}

int main() {
    float* a, * b, * out;
    float* d_a, * d_b, * d_out;

    // Allocate host memory
    a = (float*)malloc(sizeof(float) * N);
    b = (float*)malloc(sizeof(float) * N);
    out = (float*)malloc(sizeof(float) * N);

    // Initialize host arrays
    for (int i = 0; i < N; i++) {
        a[i] = 1.0f;
        b[i] = 2.0f;
    }

    // Allocate device memory
    hipMalloc((void**)&d_a, sizeof(float) * N);
    hipMalloc((void**)&d_b, sizeof(float) * N);
    hipMalloc((void**)&d_out, sizeof(float) * N);

    // Transfer data from host to device memory
    hipMemcpy(d_a, a, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(float) * N, hipMemcpyHostToDevice);

    float elapsed = 0;
    hipEvent_t start, stop;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);

    // Executing kernel 
    vector_add << <(N + 255) / 256, 256 >> > (d_out, d_a, d_b, N);


    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&elapsed, start, stop);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    printf("The elapsed time in gpu was %.2f ms", elapsed);


    // Transfer data back to host memory
    hipMemcpy(out, d_out, sizeof(float) * N, hipMemcpyDeviceToHost);

    // Verification
    for (int i = 0; i < N; i++) {
        assert(fabs(out[i] - a[i] - b[i]) < MAX_ERR);
    }
    printf("out[0] = %f\n", out[0]);
    printf("PASSED\n");

    // Deallocate device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);

    // Deallocate host memory
    free(a);
    free(b);
    free(out);
}